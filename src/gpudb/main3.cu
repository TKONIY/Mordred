#include "QueryProcessing.h"

#define HIPCUB_STDERR

#define CHECK_ERROR() { \
  hipDeviceSynchronize(); \
  hipError_t error = hipGetLastError(); \
  if(error != hipSuccess) \
  { \
    printf("CUDA error: %s\n", hipGetErrorString(error)); \
    exit(-1); \
  } \
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

tbb::task_scheduler_init init(1); // Use the default number of threads.

bool g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main () {

  CacheManager* cm = new CacheManager(1000000000, 25);

  cm->cacheColumnSegmentInGPU(cm->lo_orderdate, 6000);
  cm->cacheColumnSegmentInGPU(cm->lo_partkey, 6000);
  cm->cacheColumnSegmentInGPU(cm->lo_suppkey, 6000);
  cm->cacheColumnSegmentInGPU(cm->lo_revenue, 6000);
  cm->cacheColumnSegmentInGPU(cm->d_datekey, 3);
  cm->cacheColumnSegmentInGPU(cm->d_year, 3);
  cm->cacheColumnSegmentInGPU(cm->p_partkey, 200);
  cm->cacheColumnSegmentInGPU(cm->p_category, 200);
  cm->cacheColumnSegmentInGPU(cm->p_brand1, 200);
  cm->cacheColumnSegmentInGPU(cm->s_suppkey, 2);
  cm->cacheColumnSegmentInGPU(cm->s_region, 2);

  cm->constructListSegmentInGPU(cm->s_suppkey);
  cm->constructListSegmentInGPU(cm->s_region);
  cm->constructListSegmentInGPU(cm->p_partkey);
  cm->constructListSegmentInGPU(cm->p_category);
  cm->constructListSegmentInGPU(cm->p_brand1);
  cm->constructListSegmentInGPU(cm->d_datekey);
  cm->constructListSegmentInGPU(cm->d_year);
  cm->constructListSegmentInGPU(cm->lo_suppkey);
  cm->constructListSegmentInGPU(cm->lo_partkey);
  cm->constructListSegmentInGPU(cm->lo_orderdate);
  cm->constructListSegmentInGPU(cm->lo_revenue);

  chrono::high_resolution_clock::time_point st, finish;
  st = chrono::high_resolution_clock::now();

  int d_val_len = 19981230 - 19920101 + 1;

  int *h_ht_p = (int*)malloc(2 * P_LEN * sizeof(int));

  memset(h_ht_p, 0, 2 * P_LEN * sizeof(int));

  int *d_ht_d, *d_ht_s;
  g_allocator.DeviceAllocate((void**)&d_ht_d, 2 * d_val_len * sizeof(int));
  g_allocator.DeviceAllocate((void**)&d_ht_s, 2 * S_LEN * sizeof(int));

  hipMemset(d_ht_d, 0, 2 * d_val_len * sizeof(int));
  hipMemset(d_ht_s, 0, 2 * S_LEN * sizeof(int));

  for (int i = 0; i < 2; i++) {
    int idx_key = cm->segment_list[cm->s_suppkey->column_id][i];
    int idx_filter = cm->segment_list[cm->s_region->column_id][i];
    int* filter_col = cm->gpuCache + idx_filter * SEGMENT_SIZE;
    int* dim_key = cm->gpuCache + idx_key * SEGMENT_SIZE;
    int segment_number = i;
    //build_filter_GPU<<<((S_LEN % SEGMENT_SIZE) + 127)/128, 128>>>(filter_col, 1, dim_key, NULL, S_LEN % SEGMENT_SIZE, d_ht_s, S_LEN, 0, segment_number, 2);
    build_filter_GPU<<<(SEGMENT_SIZE + 127)/128, 128>>>(filter_col, 1, dim_key, NULL, SEGMENT_SIZE, d_ht_s, S_LEN, 0, segment_number, 1);
  }

  build_filter_CPU(cm->h_p_category, 1, cm->h_p_partkey, NULL, P_LEN, h_ht_p, P_LEN, 0, 1);

  for (int i = 0; i < 3; i++) {
    int idx_key = cm->segment_list[cm->d_datekey->column_id][i];
    int* dim_key = cm->gpuCache + idx_key * SEGMENT_SIZE;
    int segment_number = i;
    if (i == 2)
      build_GPU<<<((D_LEN % SEGMENT_SIZE) + 127)/128, 128>>>(dim_key, NULL, D_LEN % SEGMENT_SIZE, d_ht_d, d_val_len, 19920101, segment_number, 1);
    else
      build_GPU<<<(SEGMENT_SIZE + 127)/128, 128>>>(dim_key, NULL, SEGMENT_SIZE, d_ht_d, d_val_len, 19920101, segment_number, 1);
  }

  int *h_lo_off = new int[LO_LEN];
  int *h_supp_off = new int[LO_LEN];
  int *h_part_off = new int[LO_LEN];
  int *h_date_off = new int[LO_LEN];

  int *lo_off = new int[LO_LEN];
  int *supp_off = new int[LO_LEN];
  int *part_off = new int[LO_LEN];
  int *date_off = new int[LO_LEN];

  int *d_lo_off, *d_supp_off, *d_part_off, *d_date_off;
  g_allocator.DeviceAllocate((void**)&d_lo_off, LO_LEN * sizeof(int));
  g_allocator.DeviceAllocate((void**)&d_supp_off, LO_LEN * sizeof(int));
  g_allocator.DeviceAllocate((void**)&d_part_off, LO_LEN * sizeof(int));
  g_allocator.DeviceAllocate((void**)&d_date_off, LO_LEN * sizeof(int));

  int *d_res;
  int res_size = ((1998-1992+1) * (5 * 5 * 40));
  int res_array_size = res_size * 6;
  g_allocator.DeviceAllocate((void**)&d_res, res_array_size * sizeof(int));
  hipMemset(d_res, 0, res_array_size * sizeof(int));

  int offset = 0;
  int start_index = 0;

  int *total;
  int h_total = 0;
  //hipMalloc((void **)&total, 1 * sizeof(int));
  g_allocator.DeviceAllocate((void**)&total, sizeof(int));
  hipMemset(total, 0, sizeof(int));

  for (int i = 0; i < 6000; i++) {

    start_index = h_total;

    int tile_items = 128*4;
    int idx_key1 = cm->segment_list[cm->lo_suppkey->column_id][i];
    int idx_key2 = cm->segment_list[cm->lo_orderdate->column_id][i];
    int start_offset = i * SEGMENT_SIZE;

    int* dim_key1 = cm->gpuCache + idx_key1 * SEGMENT_SIZE;
    int* dim_key2 = cm->gpuCache + idx_key2 * SEGMENT_SIZE;

    probe_GPU<128,4><<<(SEGMENT_SIZE + tile_items - 1)/tile_items, 128>>> 
    (dim_key1, dim_key2, NULL, NULL,
      d_ht_s, S_LEN, d_ht_d, d_val_len, NULL, 0, NULL, 0, 0, 19920101, 0, 0, 
      d_lo_off, d_supp_off, d_date_off, NULL, NULL,
      SEGMENT_SIZE, total, start_offset);

    hipMemcpy(&h_total, total, sizeof(int), hipMemcpyDeviceToHost);

    hipMemcpy(lo_off + start_index, d_lo_off + start_index , (h_total - start_index) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(supp_off + start_index, d_supp_off + start_index , (h_total - start_index) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(date_off + start_index, d_date_off + start_index , (h_total - start_index) * sizeof(int), hipMemcpyDeviceToHost);

    // probe_2_CPU(h_t_table, cm->h_lo_partkey, h_ht_p, h_t_table_res, (h_total - start_index), P_LEN, start_index, 0, &offset);

    probe_CPU(lo_off, supp_off, date_off, NULL, NULL,
      NULL, NULL, cm->h_lo_partkey, NULL,
      NULL, 0, NULL, 0, h_ht_p, P_LEN, NULL, 0,
      0, 0, 0, 0,
      h_lo_off, h_supp_off, h_date_off, h_part_off, NULL, 
      (h_total - start_index), start_index, &offset);


    // for (int j = 0; j < offset; j++) {
    //   if (h_t_table_res[j << 2] != 0) {
    //     printf("%d %d %d %d\n", h_t_table_res[j << 2], h_t_table_res[(j << 2) + 1], h_t_table_res[(j << 2) + 2], h_t_table_res[(j << 2) + 3]);
    //   }
    // }
  }

  // int*d_lo_idx, *d_p_idx, *d_d_idx;
  // g_allocator.DeviceAllocate((void**)&d_lo_idx, cm->cache_total_seg * sizeof(int));
  // g_allocator.DeviceAllocate((void**)&d_p_idx, cm->cache_total_seg * sizeof(int));
  // g_allocator.DeviceAllocate((void**)&d_d_idx, cm->cache_total_seg * sizeof(int));

  printf("total = %d\n", offset);

  // hipMemcpy(d_lo_off, h_lo_off, offset * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_supp_off, h_supp_off, offset * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_part_off, h_part_off, offset * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_date_off, h_date_off, offset * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_lo_idx, cm->segment_list[cm->lo_revenue->column_id], cm->cache_total_seg * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_p_idx, cm->segment_list[cm->p_brand1->column_id], cm->cache_total_seg * sizeof(int), hipMemcpyHostToDevice);
  // hipMemcpy(d_d_idx, cm->segment_list[cm->d_year->column_id], cm->cache_total_seg * sizeof(int), hipMemcpyHostToDevice);

  int* res = new int[res_array_size];
  memset(res, 0, res_array_size * sizeof(int));

  runAggregationQ2CPU(cm->h_lo_revenue, cm->h_p_brand1, cm->h_d_year, h_lo_off, h_part_off, h_date_off, offset, res, res_size);

  // runAggregationQ2GPU<<<(offset + 128 - 1)/128, 128>>>(cm->gpuCache, d_lo_idx, d_p_idx, d_d_idx, d_t_table, offset, d_res, res_size);

  finish = chrono::high_resolution_clock::now();
  std::chrono::duration<double> diff = finish - st;
  // hipMemcpy(res, d_res, res_array_size * sizeof(int), hipMemcpyDeviceToHost);

  cout << "Result:" << endl;
  int res_count = 0;
  for (int i=0; i<res_size; i++) {
    if (res[6*i+1] != 0) {
      cout << res[6*i+1] << " " << res[6*i+2] << " " << reinterpret_cast<unsigned long long*>(&res[6*i+4])[0]  << endl;
      res_count += 1;
    }
  }

  cout << "Res Count: " << res_count << endl;
  cout << "Time Taken Total: " << diff.count() * 1000 << endl;

  delete cm;
  delete lo_off;
  delete supp_off;
  delete part_off;
  delete date_off;
  delete h_lo_off;
  delete h_supp_off;
  delete h_part_off;
  delete h_date_off;
  delete h_ht_p;
  delete res;

  g_allocator.DeviceFree(d_ht_s);
  g_allocator.DeviceFree(d_ht_d);
  g_allocator.DeviceFree(d_lo_off);
  g_allocator.DeviceFree(d_supp_off);
  g_allocator.DeviceFree(d_part_off);
  g_allocator.DeviceFree(d_date_off);
  // g_allocator.DeviceFree(d_lo_idx);
  // g_allocator.DeviceFree(d_p_idx);
  // g_allocator.DeviceFree(d_d_idx);

  return 0;
}



// void probe_2_CPU(int* h_t_table, int* dimkey_col1, int* ht1, int h_total, int dim_len1, int start_offset, int min_key1) {

//   // Probe
//   parallel_for(blocked_range<size_t>(0, h_total, h_total/NUM_THREADS + 4), [&](auto range) {
//     int start = range.begin();
//     int end = range.end();
//     int end_batch = start + ((end - start)/BATCH_SIZE) * BATCH_SIZE;

//     for (int batch_start = start; batch_start < end_batch; batch_start += BATCH_SIZE) {
//       #pragma simd
//       for (int i = batch_start; i < batch_start + BATCH_SIZE; i++) {
//         int hash;
//         int slot;
//         int lo_offset = h_t_table[((start_offset + i) << 2)];
//         hash = HASH(dimkey_col1[lo_offset], dim_len1, min_key1);
//         slot = ht1[hash << 1];
//         if (slot != 0) {
//           int dim_offset1 = ht1[(hash << 1) + 1];
//           h_t_table[((start_offset + i) << 2) + 3] = dim_offset1;
//         } else {
//           h_t_table[((start_offset + i) << 2)] = 0;
//           h_t_table[((start_offset + i) << 2) + 1] = 0;
//           h_t_table[((start_offset + i) << 2) + 2] = 0;
//           h_t_table[((start_offset + i) << 2) + 3] = 0;
//         }
//       }
//     }

//     for (int i = end_batch ; i < end; i++) {
//       int hash;
//       int slot;
//       int lo_offset = h_t_table[((start_offset + i) << 2)];
//       hash = HASH(dimkey_col1[lo_offset], dim_len1, min_key1);
//       slot = ht1[hash << 1];
//       if (slot != 0) {
//         int dim_offset1 = ht1[(hash << 1) + 1];
//         h_t_table[((start_offset + i) << 2) + 3] = dim_offset1;
//       } else {
//         h_t_table[((start_offset + i) << 2)] = 0;
//         h_t_table[((start_offset + i) << 2) + 1] = 0;
//         h_t_table[((start_offset + i) << 2) + 2] = 0;
//         h_t_table[((start_offset + i) << 2) + 3] = 0;
//       }
//     }
//   });