#include "hip/hip_runtime.h"
#include "QueryProcessing.h"
#include "QueryOptimizer.h"
#include "CPUGPUProcessing.h"
#include "CacheManager.h"
#include "CPUProcessing.h"
#include "CostModel.h"

// using namespace std;

int main() {
	// CPUGPUProcessing* cgp = new CPUGPUProcessing(0, 0, 0, 0, 0, 0, 1, 1, 0);
	// QueryProcessing* qp = new QueryProcessing(cgp, 0);
	// QueryOptimizer* qo = qp->qo;
	// CacheManager* cm = qo->cm;

	Normal* norm = new Normal(35, 3, 0, 315);

	for (int i = 0; i < 50; i++) {
		norm->generateNorm();
		cout << norm->date.first << " " << norm->date.second << endl;
	}

	norm->reset(300, 3);

	for (int i = 0; i < 50; i++) {
		norm->generateNorm();
		cout << norm->date.first << " " << norm->date.second << endl;
	}

	return 0;
}