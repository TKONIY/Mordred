#include "QueryProcessing.h"

#include <chrono>
#include <atomic>

bool g_verbose = false;  // Whether to display input/output to console
hipcub::CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main () {

	CacheManager* cm = new CacheManager(1000000000, 25);

  cm->cacheColumnSegmentInGPU(cm->lo_orderdate, 60);
  cm->cacheColumnSegmentInGPU(cm->lo_partkey, 60);
  cm->cacheColumnSegmentInGPU(cm->lo_suppkey, 60);
  cm->cacheColumnSegmentInGPU(cm->lo_revenue, 60);
  cm->cacheColumnSegmentInGPU(cm->d_datekey, 1);
  cm->cacheColumnSegmentInGPU(cm->d_year, 1);
  cm->cacheColumnSegmentInGPU(cm->p_partkey, 1);
  cm->cacheColumnSegmentInGPU(cm->p_category, 1);
  cm->cacheColumnSegmentInGPU(cm->p_brand1, 1);
  cm->cacheColumnSegmentInGPU(cm->s_suppkey, 1);
  cm->cacheColumnSegmentInGPU(cm->s_region, 1);

  cm->constructListSegmentInGPU(cm->s_suppkey);
  cm->constructListSegmentInGPU(cm->s_region);
  cm->constructListSegmentInGPU(cm->p_partkey);
  cm->constructListSegmentInGPU(cm->p_category);
  cm->constructListSegmentInGPU(cm->p_brand1);
  cm->constructListSegmentInGPU(cm->d_datekey);
  cm->constructListSegmentInGPU(cm->d_year);
  cm->constructListSegmentInGPU(cm->lo_suppkey);
  cm->constructListSegmentInGPU(cm->lo_partkey);
  cm->constructListSegmentInGPU(cm->lo_orderdate);
  cm->constructListSegmentInGPU(cm->lo_revenue);

  for (int trial = 0; trial < 3; trial++) {

    chrono::high_resolution_clock::time_point st, finish, bGPU1, bGPU2, pGPU1, pGPU2, gCPU1, gCPU2, tr1, tr2;
    st = chrono::high_resolution_clock::now();

    bGPU1 = chrono::high_resolution_clock::now();

  	int d_val_len = 19981230 - 19920101 + 1;

  	int *d_ht_d, *d_ht_p, *d_ht_s;
  	g_allocator.DeviceAllocate((void**)&d_ht_d, 2 * d_val_len * sizeof(int));
  	g_allocator.DeviceAllocate((void**)&d_ht_p, 2 * P_LEN * sizeof(int));
  	g_allocator.DeviceAllocate((void**)&d_ht_s, 2 * S_LEN * sizeof(int));

  	hipMemset(d_ht_d, 0, 2 * d_val_len * sizeof(int));
  	hipMemset(d_ht_p, 0, 2 * P_LEN * sizeof(int));
  	hipMemset(d_ht_s, 0, 2 * S_LEN * sizeof(int));

    for (int i = 0; i < 1; i++) {
      int idx_key = cm->segment_list[cm->s_suppkey->column_id][i];
      int idx_filter = cm->segment_list[cm->s_region->column_id][i];
      int* filter_col = cm->gpuCache + idx_filter * SEGMENT_SIZE;
      int* dim_key = cm->gpuCache + idx_key * SEGMENT_SIZE;
      int segment_number = i;
      build_filter_GPU<<<((S_LEN % SEGMENT_SIZE) + 127)/128, 128>>>(filter_col, 1, dim_key, NULL, S_LEN % SEGMENT_SIZE, d_ht_s, S_LEN, 0, segment_number, 1);
      //build_filter_GPU<<<(SEGMENT_SIZE + 127)/128, 128>>>(filter_col, 1, dim_key, NULL, SEGMENT_SIZE, d_ht_s, S_LEN, 0, segment_number, 1);
    }

    for (int i = 0; i < 1; i++) {
      int idx_key = cm->segment_list[cm->p_partkey->column_id][i];
      int idx_filter = cm->segment_list[cm->p_category->column_id][i];
      int* filter_col = cm->gpuCache + idx_filter * SEGMENT_SIZE;
      int* dim_key = cm->gpuCache + idx_key * SEGMENT_SIZE;
      int segment_number = i;
      build_filter_GPU<<<((P_LEN % SEGMENT_SIZE + 127))/128, 128>>>(filter_col, 1, dim_key, NULL, P_LEN % SEGMENT_SIZE, d_ht_p, P_LEN, 0, segment_number, 1);
    }

    for (int i = 0; i < 1; i++) {
      int idx_key = cm->segment_list[cm->d_datekey->column_id][i];
      int* dim_key = cm->gpuCache + idx_key * SEGMENT_SIZE;
      int segment_number = i;
      if (i == 0)
        build_GPU<<<((D_LEN % SEGMENT_SIZE) + 127)/128, 128>>>(dim_key, NULL, D_LEN % SEGMENT_SIZE, d_ht_d, d_val_len, 19920101, segment_number, 1);
      else
        build_GPU<<<(SEGMENT_SIZE + 127)/128, 128>>>(dim_key, NULL, SEGMENT_SIZE, d_ht_d, d_val_len, 19920101, segment_number, 1);
    }

    bGPU2 = chrono::high_resolution_clock::now();
    std::chrono::duration<double> buildtimeGPU = bGPU2 - bGPU1;

    pGPU1 = chrono::high_resolution_clock::now();    

    int *d_lo_off, *d_supp_off, *d_part_off, *d_date_off;
    g_allocator.DeviceAllocate((void**)&d_lo_off, 500000 * sizeof(int));
    g_allocator.DeviceAllocate((void**)&d_supp_off, 500000 * sizeof(int));
    g_allocator.DeviceAllocate((void**)&d_part_off, 500000 * sizeof(int));
    g_allocator.DeviceAllocate((void**)&d_date_off, 500000 * sizeof(int));

    int *total;
    int h_total;
    hipMalloc((void **)&total, sizeof(int));
    hipMemset(total, 0, sizeof(int));

    int res_size = (1998-1992+1) * 5 * 5 * 40;
    int res_array_size = res_size * 6;
    int* res = new int[res_array_size];

    memset(res, 0, res_array_size * sizeof(int));

    for (int i = 0; i < 60; i++) {
      int tile_items = 128*4;
      int idx_key1 = cm->segment_list[cm->lo_suppkey->column_id][i];
      int idx_key2 = cm->segment_list[cm->lo_partkey->column_id][i];
      int idx_key3 = cm->segment_list[cm->lo_orderdate->column_id][i];
      int start_offset = i * SEGMENT_SIZE;

      int* dim_key1 = cm->gpuCache + idx_key1*SEGMENT_SIZE;
      int* dim_key2 = cm->gpuCache + idx_key2*SEGMENT_SIZE;
      int* dim_key3 = cm->gpuCache + idx_key3*SEGMENT_SIZE;

      if (i == 59) {
        probe_GPU<128,4><<<((LO_LEN % SEGMENT_SIZE) + tile_items - 1)/tile_items, 128>>>
        (dim_key1, dim_key2, dim_key3, NULL, d_ht_s, S_LEN, d_ht_p, P_LEN, d_ht_d, d_val_len, NULL, 0,
          0, 0, 19920101, 0, d_lo_off, d_supp_off, d_part_off, d_date_off, NULL, 
          (LO_LEN % SEGMENT_SIZE), total, start_offset);    
      } else {
        probe_GPU<128,4><<<(SEGMENT_SIZE + tile_items - 1)/tile_items, 128>>>
        (dim_key1, dim_key2, dim_key3, NULL, d_ht_s, S_LEN, d_ht_p, P_LEN, d_ht_d, d_val_len, NULL, 0,
          0, 0, 19920101, 0, d_lo_off, d_supp_off, d_part_off, d_date_off, NULL, 
          SEGMENT_SIZE, total, start_offset);
      }
    }

    pGPU2 = chrono::high_resolution_clock::now();
    std::chrono::duration<double> probetimeGPU = pGPU2 - pGPU1;

    tr1 = chrono::high_resolution_clock::now();

    hipMemcpy(&h_total, total, sizeof(int), hipMemcpyDeviceToHost);

    int *h_lo_off = new int[h_total];
    int *h_part_off = new int[h_total];
    int *h_supp_off = new int[h_total];
    int *h_date_off = new int[h_total];

    hipMemcpy(h_lo_off, d_lo_off, h_total * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_part_off, d_part_off, h_total * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_supp_off, d_supp_off, h_total * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_date_off, d_date_off, h_total * sizeof(int), hipMemcpyDeviceToHost);

    tr2 = chrono::high_resolution_clock::now();
    std::chrono::duration<double> transfertime = tr2 - tr1;

    gCPU1 = chrono::high_resolution_clock::now();

    printf("total = %d\n", h_total);

    runAggregationQ2CPU(cm->h_lo_revenue, cm->h_p_brand1, cm->h_d_year, h_lo_off, h_part_off, h_date_off, h_total, res, res_size);

    gCPU2 = chrono::high_resolution_clock::now();
    std::chrono::duration<double> groupbytimeCPU = gCPU2 - gCPU1;

    finish = chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = finish - st;

    cout << "Result:" << endl;
    int res_count = 0;
    for (int i=0; i<res_size; i++) {
      if (res[6*i+1] != 0) {
        cout << res[6*i+1] << " " << res[6*i+2] << " " << reinterpret_cast<unsigned long long*>(&res[6*i+4])[0]  << endl;
        res_count += 1;
      }
    }

    cout << "Res Count: " << res_count << endl;
    cout << "Time Taken Total: " << diff.count() * 1000 << endl;
    cout << "Build GPU Time Taken Total: " << buildtimeGPU.count() * 1000 << endl;
    cout << "Probe GPU Time Taken Total: " << probetimeGPU.count() * 1000 << endl;
    cout << "Group By CPU Time Taken Total: " << groupbytimeCPU.count() * 1000 << endl;
    cout << "Transfer Time Taken Total: " << transfertime.count() * 1000 << endl;

    delete h_lo_off;
    delete h_part_off;
    delete h_date_off;
    delete res;

    g_allocator.DeviceFree(d_ht_p);
    g_allocator.DeviceFree(d_ht_s);
    g_allocator.DeviceFree(d_ht_d);
    g_allocator.DeviceFree(d_lo_off);
    g_allocator.DeviceFree(d_supp_off);
    g_allocator.DeviceFree(d_part_off);
    g_allocator.DeviceFree(d_date_off);

  }

	delete cm;

	return 0;
}